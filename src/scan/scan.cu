#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

constexpr int ThreadPerBlock = 2;
constexpr int ElementPerBlock = ThreadPerBlock * 2;

__global__ void scan_block(int* a, int* output, int *block_sums, int n) {
    // load
    __shared__ int temp[ElementPerBlock];
    int idx = blockDim.x * blockIdx.x + threadIdx.x; //2
    int tid = threadIdx.x; //0
    int bid = blockIdx.x;
    if (2 * idx < n) {
        temp[2 * tid] = a[2 * idx]; 
    }
    if (2 * idx + 1 < n) {
        temp[2 * tid + 1] = a[2 * idx + 1];
    }

    int t = ElementPerBlock >> 1;
    for (int s = 1; s < ElementPerBlock; s *= 2) {
        __syncthreads();
        if (tid < t) {
            int k = tid * 2 * s;
            int i = k + s - 1; 
            int j = k + 2 * s - 1;

            temp[j] += temp[i];
        }
        t >>= 1;
    }

    if (tid == 0) {
        block_sums[bid] = temp[ElementPerBlock - 1];
        temp[ElementPerBlock - 1] = 0;
    }
    t = 1;
    for (int s = ElementPerBlock >> 1; s > 0; s >>= 1) {
        __syncthreads();
        if (tid < t) {
            int k = tid * 2 * s;
            int i = k + s - 1; 
            int j = k + 2 * s - 1;

            int tt = temp[j];
            temp[j] += temp[i];
            temp[i] = tt; 
        }
        t *= 2;
    }

    __syncthreads();

    if (2 * idx < n) {
        output[2 * idx] =  temp[2 * tid]; 
    }
    if (2 * idx + 1 < n) {
        output[2 * idx + 1] = temp[2 * tid + 1];
    }
}



__global__ void add_kernel(int *output, int *sums, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;

    if (2 * idx < n) {
        output[2 * idx] += sums[bid];
    }
    if (2 * idx + 1 < n) {
        output[2 * idx + 1] += sums[bid];
    }
}





void scan_large(int *input, int *output, int n) {
    int *d_input;
    int *d_output;
    int *d_sums;
    int *d_sums_sums;
    
    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));

    hipMemcpy(d_input, input, n * sizeof(int), hipMemcpyHostToDevice);

    int numBlock = (n + ElementPerBlock - 1) / ElementPerBlock;

    hipMalloc(&d_sums, numBlock * sizeof(int));
    hipMalloc(&d_sums_sums, numBlock * sizeof(int));

    {
        hipEvent_t start, stop;
        float elapsedTime = 0.0;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        scan_block<<<numBlock, ThreadPerBlock>>>(d_input, d_output, d_sums, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("scan_large time: %f ms\n", elapsedTime);
    }
    if (numBlock != 1) {
        scan_large(d_sums, d_sums_sums, numBlock);
        hipEvent_t start, stop;
        float elapsedTime = 0.0;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        add_kernel<<<numBlock, ThreadPerBlock>>>(d_output, d_sums_sums, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("add_kernel time: %f ms\n", elapsedTime);
    } 

    hipMemcpy(output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);
}



int main() {
    int input[] = {1, 2, 3, 0, 1, 1, 1, 1, 1};
    int output[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
    scan_large(input, output, 9);
    for (int i = 0; i < 9; i++) {
        std::cout << output[i] << ',';
    }
    std::cout << std::endl;
}